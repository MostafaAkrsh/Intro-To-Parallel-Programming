#include "hip/hip_runtime.h"
// Homework 1
// Color to Greyscale Conversion

//A common way to represent color images is known as RGBA - the color
//is specified by how much Red, Grean and Blue is in it.
//The 'A' stands for Alpha and is used for transparency, it will be
//ignored in this homework.

//Each channel Red, Blue, Green and Alpha is represented by one byte.
//Since we are using one byte for each color there are 256 different
//possible values for each color.  This means we use 4 bytes per pixel.

//Greyscale images are represented by a single intensity value per pixel
//which is one byte in size.

//To convert an image from color to grayscale one simple method is to
//set the intensity to the average of the RGB channels.  But we will
//use a more sophisticated method that takes into account how the eye 
//perceives color and weights the channels unequally.

//The eye responds most strongly to green followed by red and then blue.
//The NTSC (National Television System Committee) recommends the following
//formula for color to greyscale conversion:

//I = .299f * R + .587f * G + .114f * B

//Notice the trailing f's on the numbers which indicate that they are 
//single precision floating point constants and not double precision
//constants.

//You should fill in the kernel as well as set the block and grid sizes
//so that the entire image is processed.

#include "utils.h"
#include "hip/hip_runtime.h"
#include <>

#define BLOCK_SIZE 32

__global__
void rgba_to_greyscale(const uchar4* const rgbaImage,
                       unsigned char* const greyImage,
                       int numRows, int numCols)
{
    //Calculate the image index
    int iX = blockIdx.x * blockDim.x + threadIdx.x;
    int iY = blockIdx.y * blockDim.y + threadIdx.y;

    //Check that the thread within the image 
    if (iX >= numCols || iY >= numRows) return ;

    int index = iY * numCols + iX;
    uchar4 tempPixel = rgbaImage[index];

    greyImage[index] = (unsigned char)(.299f * tempPixel.x + .587f * tempPixel.y + .114f * tempPixel.z);
   
}

void your_rgba_to_greyscale(const uchar4 * const h_rgbaImage, uchar4 * const d_rgbaImage,
                            unsigned char* const d_greyImage, size_t numRows, size_t numCols)
{
  //You must fill in the correct sizes for the blockSize and gridSize
  //currently only one block with one thread is being launched
  const dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE, 1); 

  unsigned int gridSizeX = (unsigned int)(numCols / (BLOCK_SIZE));
  unsigned int gridSizeY = (unsigned int)(numRows / (BLOCK_SIZE));

  const dim3 gridSize(gridSizeX, gridSizeY, 1);  

  rgba_to_greyscale<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);
  
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

}
